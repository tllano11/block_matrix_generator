
#include <hip/hip_runtime.h>
#include<iostream>
#include<assert.h>

__global__ void multiply(double* m, double* v, double* r,
		       uint32_t rows, uint32_t cols) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < rows) {
    uint32_t begin = idx*rows;
    uint32_t end = begin + cols;
    uint32_t j = 0;

    for (uint32_t i = begin; i < end; ++i, ++j) {
      r[idx] += m[i] * v[j];
    }
  }
}

int main() {
  double a[4] = {1, 1, 1, 1};
  double x[2] = {1, 1};
  double b[2] = {0, 0};
  uint32_t rows = 2;
  uint32_t cols = 2;

  double *a_g, *x_g, *b_g;

  assert(hipSuccess == hipMalloc((void **) &a_g, rows*cols*sizeof(double)));
  assert(hipSuccess == hipMalloc((void **) &x_g, rows*sizeof(double)));
  assert(hipSuccess == hipMalloc((void **) &b_g, rows*sizeof(double)));

  assert(hipSuccess == hipMemcpy(a_g, a, rows*cols*sizeof(double), hipMemcpyHostToDevice));
  assert(hipSuccess == hipMemcpy(x_g, x, rows*sizeof(double), hipMemcpyHostToDevice));
  assert(hipSuccess == hipMemcpy(b_g, b, rows*sizeof(double), hipMemcpyHostToDevice));

  int tpb = 32;
  int bpg = rows*cols + (tpb - 1) / tpb;
  multiply <<< bpg, tpb  >>> (a, x, b, rows, cols);

  assert(hipSuccess == hipMemcpy(b, b_g, rows*sizeof(double), hipMemcpyDeviceToHost));

  std::cout << b[0] << b[1]  << "\n";

}
